// Implementation of the CJP algorithm, proposed by Nguyen Quang Anh Pham and Rui Fan, for parallel graph coloring on GPUs.
// Copyright (C) 2020, Cem Sakızcı <sakizcicem@gmail.com>

// This file is part of Implementing-graphColoring-on-GPU.

// Implementing-graphColoring-on-GPU is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// Implementing-graphColoring-on-GPU is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with Implementing-graphColoring-on-GPU.  If not, see <http://www.gnu.org/licenses/>.


#include "graphColoring_GPU.cuh" 


extern void graphColoring_GPU(int numberOfVertices, Vertex *vertices){
		
    hipEvent_t start,  //!< CUDA Timer start.
        		stop;   //!< CUDA Timer stop.
    hipEventCreate(&start);
	hipEventCreate(&stop);
	float total_elapsed_time, milliseconds = 0;
	int max_color = 0;
	while(true) {
		hipError_t error;
		int numberOfVerticesToBeColored(0);  //!< Number of the vertices that should be colored.
		// In this for loop, the number of vertices, having count value as 0, is determined.
		for(int i=0; i<numberOfVertices; i++) {
			if(vertices[i].count == 0 && vertices[i].color == -1) 
				numberOfVerticesToBeColored++;
		}
		//printf("# VERTICES TO BE COLORED : %d\n", numberOfVerticesToBeColored); // UNCOMMENT if you need to see this additional information.

		// Checking if all vertices have been colored.
		if(numberOfVerticesToBeColored == 0) {
			break;
		}

		Vertex *verticesToBeColored; //!< Vertices to be colored in vertex form.

		int *h_neighborSizeArray,    //!< Array that contains size of the neighbors.
			*h_colors_found;    	 //!< Array that contains the colors found.

		// Allocation on host memory for vertices that will be colored.
		verticesToBeColored = (Vertex *) malloc(sizeof(Vertex) * numberOfVerticesToBeColored);
		// Allocation on host memory for neighbor sizes of vertices(INPUT).
		h_neighborSizeArray  = (int *) malloc(sizeof(int) * numberOfVerticesToBeColored);
		// Memory allocation on host for the colors(OUTPUT).
		h_colors_found = (int *) malloc(sizeof(int) * numberOfVerticesToBeColored);

		// Checking if there were any failures while allocating host data.
		if(h_neighborSizeArray == NULL || h_colors_found == NULL) {
			fprintf(stderr, "Failed to allocate host data!\n");
			exit(EXIT_FAILURE);
		}

		// Initializing h_colors_found with -1.
		for(int i=0; i<numberOfVerticesToBeColored; i++) {
			h_colors_found[i] = -1;
		}

		// Setting verticesToBeColored and h_neighborSizeArray.
		int totalNumberOfNeighbors = 0, individualNeighborNumber = 0;
		for(int i=0, j=0; i<numberOfVertices; i++) {
			if(vertices[i].count == 0 && vertices[i].color == -1) {
				verticesToBeColored[j] = vertices[i];
				individualNeighborNumber = vertices[i].arraySize;
				h_neighborSizeArray[j] = individualNeighborNumber;
				totalNumberOfNeighbors += individualNeighborNumber; //Accumulating the neighbor size of each vertex to obtain the total for memory allocation. 
				j++;
			}
		}

		// Memory allocation on host for all neighbors of all vertices(INPUT).
		int neighborsSizeInBytes = sizeof(Vertex) * totalNumberOfNeighbors;
		Vertex *h_neighborsOfAllVertices = (Vertex *) malloc(neighborsSizeInBytes);

		// Checking if there were any failures while allocating host data.
		if(h_neighborsOfAllVertices == NULL) {
			fprintf(stderr, "Failed to allocate host data!\n");
			exit(EXIT_FAILURE);
		}

		// Setting h_neighborsOfAllVertices. 
		int cumulativeIndex = 0;
		for(int i=0; i<numberOfVerticesToBeColored; i++) {
			int neighborSize = h_neighborSizeArray[i];
			for(int j=0; j<neighborSize; j++) {
				h_neighborsOfAllVertices[cumulativeIndex] = vertices[verticesToBeColored[i].neighboursIndices[j]];
				cumulativeIndex++;
			}
		}

		// Device allocation of all neighbors(INPUT).
		Vertex *d_neighborsOfAllVertices = NULL;
		error = hipMalloc((void **) &d_neighborsOfAllVertices, neighborsSizeInBytes);
			
		if(error != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_neighborsOfAllVertices (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		
		// Device allocation of neighbor sizes(INPUT).
		int *d_neighborSizeArray = NULL;
		error = hipMalloc((void **) &d_neighborSizeArray, numberOfVerticesToBeColored * sizeof(int));
			
		if(error != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_neighborSizeArray (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		// Device allocation of colors_found(OUTPUT).
		int *d_colors_found = NULL;
		error = hipMalloc((void **) &d_colors_found, numberOfVerticesToBeColored * sizeof(int));
			
		if(error != hipSuccess) {
			fprintf(stderr, "Failed to allocate d_colors_found (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}
		
		// Copying of colors from host to device.
		//printf("Copy colors from the host memory to the CUDA device\n"); // UNCOMMENT if you need to see this additional information.
		error = hipMemcpy(d_colors_found, h_colors_found, numberOfVerticesToBeColored * sizeof(int), hipMemcpyHostToDevice);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to copy h_colors_found to device (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		//Copying of all neighbors from host to device.
		//printf("Copy all neighbors from the host memory to the CUDA device\n"); // UNCOMMENT if you need to see this additional information.
		error = hipMemcpy(d_neighborsOfAllVertices, h_neighborsOfAllVertices, neighborsSizeInBytes, hipMemcpyHostToDevice);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to copy h_neighborsOfAllVertices to device (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		//Copying of neighborSizeArray from host to device.
		//printf("Copy neighborSizeArray from the host memory to the CUDA device\n"); // UNCOMMENT if you need to see this additional information.
		error = hipMemcpy(d_neighborSizeArray, h_neighborSizeArray, numberOfVerticesToBeColored * sizeof(int), hipMemcpyHostToDevice);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to copy h_neighborSizeArray to device (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		int current_phase = 0;
		int threads_per_block = 32;
		// Here, the current phase is incremented by 1 until all vertices to be colored find their appropriate colors.
		while(true) {

			//printf("CUDA kernel launch in %d.phase\n", current_phase); // UNCOMMENT if you need to see this additional information.
			hipEventRecord(start);
			find_the_color<<<numberOfVerticesToBeColored, threads_per_block>>>(d_neighborsOfAllVertices, d_neighborSizeArray, current_phase, d_colors_found);
			hipEventRecord(stop);
			hipEventElapsedTime(&milliseconds, start, stop);
			total_elapsed_time += 1000.0f * (milliseconds);

			error = hipGetLastError();

			if(error != hipSuccess) {
				fprintf(stderr, "Failed to launch the kernel (error code %s)!\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}

			// Copy d_colors_found to host.
			//printf("Copy d_colors_found from the CUDA device to the host memory\n"); // UNCOMMENT if you need to see this additional information.
			error = hipMemcpy(h_colors_found, d_colors_found, sizeof(int) * numberOfVerticesToBeColored, hipMemcpyDeviceToHost);

			if(error != hipSuccess) {
				fprintf(stderr, "Failed to copy d_colors_found from device to host (error code %s)!\n", hipGetErrorString(error));
				exit(EXIT_FAILURE);
			}

			//printf("Copying has successfully completed\n"); // UNCOMMENT if you need to see this additional information.

			// Finding if any vertex from verticesToBeColored array could not be colored for the current phase.
			int numberOfUncoloredVertices = 0;
			for(int i=0; i<numberOfVerticesToBeColored; i++) {
				if(h_colors_found[i] != -1) {
					if(h_colors_found[i] >= 256*current_phase+1 && h_colors_found[i] <= 256*(current_phase+1)) {
						int index = verticesToBeColored[i].vertexIndex;
						int color = h_colors_found[i];
						//printf("color of V[%d] = %d\n", index, color); // UNCOMMENT if you need to see this additional information.
						vertices[index].color = color; // setting the color.
						// To find the maximum color used.
						if(color > max_color) {
							max_color = color;
						}

						// Decrement neighbor's count values by 1.
						int neighborSize = vertices[index].arraySize;
						for(int j=0; j<neighborSize; j++) {
							//printf("N_id: %d, N_color: %d\n",vertices[index].neighboursIndices[j], vertices[vertices[index].neighboursIndices[j]].color); // UNCOMMENT if you need to see this additional information.
							if(vertices[vertices[index].neighboursIndices[j]].count > 0)
								vertices[vertices[index].neighboursIndices[j]].count--;
						}
					}
					
				}
				else {
					numberOfUncoloredVertices++;
				}
			}	
			
			if(numberOfUncoloredVertices > 0) {
				
				printf("Going to the next phase...\n");
				current_phase++;
				
			}
			else {
				break;
			}
			
		}
		// Free device global memory
		error = hipFree(d_neighborsOfAllVertices);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to free d_neighborsOfAllVertices (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		error = hipFree(d_neighborSizeArray);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to free d_neighborSizeArray (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		error = hipFree(d_colors_found);

		if(error != hipSuccess) {
			fprintf(stderr, "Failed to free d_colors_found (error code %s)!\n", hipGetErrorString(error));
			exit(EXIT_FAILURE);
		}

		//printf("DELETED ALLOCATED MEMORY FOR DEVICE\n"); // UNCOMMENT if you need to see this additional information.

		// Free host memory
		free(h_neighborsOfAllVertices);
		free(h_neighborSizeArray);
		free(h_colors_found);
		//printf("DELETED ALLOCATED MEMORY FOR HOST \n"); // UNCOMMENT if you need to see this additional information.

	}
	
}
